#include "hip/hip_runtime.h"
#include "lodepng.h"
// #include "hip/hip_runtime_api.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <algorithm>
#include <stdio.h>
#include <time.h>

#define TILE_W  16
#define TILE_H  16
#define R        2
#define D       (R*2+1)
#define S    (D*D)
#define BLOCK_W (TILE_W+(2*R))
#define BLOCK_H (TILE_H+(2*R))

#define CHANNELS 3

__device__ 
void sort(unsigned char* input){
	for(int i = 0; i < 8; i++){
		int iMin = i;

		for(int j = i+1; j < 9; j++){
			if(input[i] < input[iMin]){
				iMin = j;
			}
		}
		
		if(iMin != i){
			unsigned char temp = input[i];
			input[i] = input[iMin];
			input[iMin] = temp;
		}
	}
}

__global__
void mirror(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    
//    int col = 3 * (blockIdx.x * blockDim.x + threadIdx.x);
//    int row = 3 * (blockIdx.y * blockDim.y + threadIdx.y);
//
//    if ( row >= width || col >= height ) { return; }
//
//    int col_new = col;
//    int row_new = width - row;
//
//    int myId = row * height + col;
//    int myId_new = row_new * height + col_new;
//
//    output_image[myId_new] = input_image[myId];
    
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    /* Check if Offset is Within Bounds */
    if (offset < width * height) {
        
        const int currentoffset = offset * 3;
        
        /* Get Current Color Values */
        float output_red = input_image[currentoffset];
        float output_green = input_image[currentoffset + 1];
        float output_blue = input_image[currentoffset + 2];
        
        /* Assign Inverted Color Values */
        output_image[offset * 3] = output_red;
        output_image[offset * 3 + 1] = output_green;
        output_image[offset * 3 + 2] = output_blue;
    }
}

__global__
void invert(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    /* Check if Offset is Within Bounds */
    if (offset < width * height) {
        
        const int currentoffset = offset * 3;
        
        /* Get Current Color Values */
        float output_red = input_image[currentoffset];
        float output_green = input_image[currentoffset + 1];
        float output_blue = input_image[currentoffset + 2];
        
        /* Assign Inverted Color Values */
        output_image[offset * 3] = 255 - output_red;
        output_image[offset * 3 + 1] = 255 - output_green;
        output_image[offset * 3 + 2] = 255 - output_blue;
    }
}

__global__
void h_average(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    /* Check if Offset is Within Bounds */
    if (offset < width * height) {
        
        const int currentoffset = offset * 3;
        
        /* Get Current Color Values */

        float output_red, output_green, output_blue;

        if(offset > 0 && offset < width*height - 1) {
            float output_red = (input_image[currentoffset] + input_image[(offset-1)*3] + input_image[(offset+1)*3])/3;
            float output_green = (input_image[currentoffset + 1] + input_image[(offset-1)*3 + 1] + input_image[(offset+1)*3 + 1])/3;
            float output_blue = (input_image[currentoffset + 2] + input_image[(offset-1)*3 + 2] + input_image[(offset+1)*3 + 2])/3;  
        }
        else {
            float output_red = input_image[currentoffset];
            float output_green = input_image[currentoffset + 1];
            float output_blue = input_image[currentoffset + 2];  
        }


        
        
        /* Assign Inverted Color Values */
        output_image[offset * 3] = 255 - output_red;
        output_image[offset * 3 + 1] = 255 - output_green;
        output_image[offset * 3 + 2] = 255 - output_blue;
    }
}

__global__
void greyscale(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    
    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    /* Check if Offset is Within Bounds */
    if (offset < width * height) {
        
        const int currentoffset = offset * 3;
        
        /* Get Current Color Values */
        float output_red = 0.21 * input_image[currentoffset];
        float output_green = 0.72 * input_image[currentoffset + 1];
        float output_blue = 0.07 * input_image[currentoffset + 2];
        float output_color = output_red + output_green + output_blue;
        
        /* Assign Inverted Color Values */
        output_image[offset * 3] = output_color;
        output_image[offset * 3 + 1] = output_color;
        output_image[offset * 3 + 2] = output_color;
    }
}

__global__ 
void simple_filter(int *input_image, int *g_odata, unsigned int width, unsigned int height){
    __shared__ int smem[BLOCK_W*BLOCK_H];
    int x = blockIdx.x*TILE_W + threadIdx.x - R;
    int y = blockIdx.y*TILE_H + threadIdx.y - R;

    x = max(0, x);
    x = min(x, width-1);
    y = max(y, 0);
    y = min(y, height-1);

    unsigned int index = y*width + x;
    unsigned int bindex = threadIdx.y*blockDim.y+threadIdx.x;

    smem[bindex] = input_image[index];
    __syncthreads();

    if((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W-R)) && (threadIdx.y >= R) && (threadIdx.y < (BLOCK_H-R))){
        float sum = 0;
        for(int dy = -R; dy <= R; dy++){
            for(int dx = -R; dx <= R; dx++){
                float i = smem[bindex + (dy*blockDim.x) + dx];
                sum += i;
            }
        }
        g_odata[index] = sum/S;
    }
}

__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {
    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 3; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}

__global__ void
medianFilter(unsigned char* input_image, unsigned char* output_image, int width, int height){

	const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
	int x = offset % width;
	int y = (offset - x)/width;

	if(offset < width*height){
	
		unsigned char filterVectorRed[9] = {0,0,0,0,0,0,0,0,0};
		unsigned char filterVectorGreen[9] = {0,0,0,0,0,0,0,0,0};
		unsigned char filterVectorBlue[9] = {0,0,0,0,0,0,0,0,0};

		if(y == 0 || y == height - 1 || x == 0 || x == width - 1){
			output_image[offset*3] = input_image[offset];
			output_image[offset*3 + 1] = input_image[offset + 1];
			output_image[offset*3 + 2] = input_image[offset + 2];
		}
		else{
			int i = 0;
			for(int dx = -1; dx <= 1; dx++){
				for(int dy = -1; dy <= 1; dy++){
					if(x+dx >= 0 && x+dx < width && y+dy >= 0 && y+dy < height){
						const int currentOffset = (offset+dx+dy*width)*3;
						filterVectorRed[i] = input_image[currentOffset];
						filterVectorGreen[i] = input_image[currentOffset + 1];
						filterVectorBlue[i] = input_image[currentOffset + 2];
						i++;
					}
				}
			}
			sort(filterVectorRed);
			sort(filterVectorGreen);		
			sort(filterVectorBlue);

			output_image[offset*3] = filterVectorRed[4];
			output_image[offset*3 + 1] = filterVectorGreen[4];
			output_image[offset*3 + 2] = filterVectorBlue[4];
		}
	}

}

__device__ float exp(int i) { return exp((float) i); }

const int BLOCKDIM = 32;
const int sigma1 = 50;
const int sigma2 = 50;

__device__ const int FILTER_SIZE = 9;
__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__global__ 
void bilateral_filter_2d(unsigned char* input, unsigned char* output, int width, int height)
{
    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;

	if(offset < width*height) {
		float running_total = 0;
		float norm_factor = 0;
		const int offset = y * width + x;
		for (int xctr = -FILTER_HALFSIZE; xctr <= FILTER_HALFSIZE; xctr++) 
		{
			for (int yctr = -FILTER_HALFSIZE; yctr <= FILTER_HALFSIZE; yctr++) 
			{
				int y_iter = y + xctr;
				int x_iter = x + yctr;
				if (x_iter < 0) x_iter = -x_iter;
				if (y_iter < 0) y_iter = -y_iter;
				if (x_iter > width-1) x_iter = width-1-xctr;
				if (y_iter > height-1) y_iter = height-1-yctr;
				float intensity_change = input[y_iter * width + x_iter] - input[y * width + x];
				float w1 = exp(-(xctr * xctr + yctr * yctr) / (2 * sigma1 * sigma1));
				float w2 = exp(-(intensity_change * intensity_change) / (2 * sigma2 * sigma2));
				running_total += input[y_iter * width + x_iter] * w1 * w2;
				norm_factor += w1 * w2;
			}
		}
        output[offset] = running_total / norm_factor;
        
	}
}

void getError(hipError_t err) {
    if(err != hipSuccess) {
        std::cout << "Error " << hipGetErrorString(err) << std::endl;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height, char* arg) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice ));
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

    /* Dimensions */
    dim3 blockDims(512, 1, 1);
    dim3 gridDims((unsigned int) ceil((double)(width*height * 3 / blockDims.x)), 1, 1 );

    // timet_t start, end;
    // start = clock();
    // end = clock();
    // std::cout << "Blur Filter took " << (end-start)/CLOCKS_PER_SEC << " ms\n";
    
    switch (arg[0]) {
        
        /* Blur */
        case 'b':
        case 'B':
            blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            
        /* Greyscale */
        case 'g':
        case 'G':
            greyscale<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
            
        /* Invert */
        case 'i':
        case 'I':
            invert<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
        
        /* Median */
        case 'm':
        case 'M':
            medianFilter<<<gridDims, blockDims>>>(dev_input, dev_output, width, height);
        
        /* Invalid Argument */
        default:
            printf("Invalid Argument. Options are: b, g, i, m\n"); exit(1);
    
    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}


int main(int argc, char *argv[]){
    // std::cout << argv[1];

    const char* input_file = argv[1];
    const char* output_file = argv[2];

    std::vector<unsigned char> in_image;
    unsigned int width, height;

    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    if(error) std::cout << "decoder error " << error << ": " << lodepng_error_text(error) << std::endl;

    // Prepare the data
    unsigned char* input_image = new unsigned char[(in_image.size()*3)/4];
    unsigned char* output_image = new unsigned char[(in_image.size()*3)/4];
    int where = 0;
    for(int i = 0; i < in_image.size(); ++i) {
       if((i+1) % 4 != 0) {
           input_image[where] = in_image.at(i);
           output_image[where] = 255;
           where++;
       }
    }

    // Run the filter on it
    if (argc < 4) {
        printf("Invalid Usage\n");
        printf("Command should be of the form: ./filter input_image.png output_image.png <b, g, i, m>\n");
        exit(1);
    }
    else { filter(input_image, output_image, width, height, argv[3]); }

    // Prepare data for output
    std::vector<unsigned char> out_image;
    for(int i = 0; i < in_image.size(); ++i) {
        out_image.push_back(output_image[i]);
        if((i+1) % 3 == 0) {
            out_image.push_back(255);
        }
    }
    
    // Output the data
    error = lodepng::encode(output_file, out_image, width, height);

    //if there's an error, display it
    if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;

    delete[] input_image;
    delete[] output_image;
    return 0;

}
