#include "hip/hip_runtime.h"
#include "lodepng.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <algorithm>
#include <stdio.h>


#define TILE_W  16
#define TILE_H  16
#define R        2
#define D       (R*2+1)
#define S    (D*D)
#define BLOCK_W (TILE_W+(2*R))
#define BLOCK_H (TILE_H+(2*R))

#define CHANNELS 3

// __global__ void d_filter(int *g_idata, int *g_odata, unsigned int width, unsigned int height){
//     __shared__ int smem[BLOCK_W*BLOCK_H];
//     int x = blockIdx.x*TILE_W + threadIdx.x - R;
//     int y = blockIdx.y*TILE_H + threadIdx.y - R;

//     x = max(0, x);
//     x = min(x, width-1);
//     y = max(y, 0);
//     y = min(y, height-1);

//     unsigned int index = y*width + x;
//     unsigned int bindex = threadIdx.y*blockDim.y+threadIdx.x;

//     smem[bindex] = g_idata[index];
//     __syncthreads();

//     if((threadIdx.x >= R) && (threadIdx.x < (BLOCK_W-R)) && (threadIdx.y >= R) && (threadIdx.y < (BLOCK_H-R))){
//         float sum = 0;
//         for(int dy = -R; dy <= R; dy++){
//             for(int dx = -R; dx <= R; dx++){
//                 float i = smem[bindex + (dy*blockDim.x) + dx];
//                 sum += i;
//             }
//         }
//         g_odata[index] = sum/S;
//     }
// }

__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
        }
}

// __global__ void colorConvert(unsigned char * rgbImage, unsigned char * grayImage, int width, int height) {
//     const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
//     int x = offset % width;
//     int y = (offset-x)/width;

//     }
// }


void getError(hipError_t err) {
    if(err != hipSuccess) {
        std::cout << "Error " << hipGetErrorString(err) << std::endl;
    }
}

void filter (unsigned char* input_image, unsigned char* output_image, int width, int height) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
    getError(hipMemcpy( dev_input, input_image, width*height*3*sizeof(unsigned char), hipMemcpyHostToDevice ));
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );

    colorConvert<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 


    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}


int main(int argc, char *argv[]){
    // std::cout << argv[1];

    const char* input_file = argv[1];
    const char* output_file = argv[2];

    std::vector<unsigned char> in_image;
    unsigned int width, height;

    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    if(error) std::cout << "decoder error " << error << ": " << lodepng_error_text(error) << std::endl;

    // Prepare the data
    unsigned char* input_image = new unsigned char[(in_image.size()*3)/4];
    unsigned char* output_image = new unsigned char[(in_image.size()*3)/4];
    int where = 0;
    for(int i = 0; i < in_image.size(); ++i) {
       if((i+1) % 4 != 0) {
           input_image[where] = in_image.at(i);
           output_image[where] = 255;
           where++;
       }
    }

    // Run the filter on it
    filter(input_image, output_image, width, height); 

    // Prepare data for output
    std::vector<unsigned char> out_image;
    for(int i = 0; i < in_image.size(); ++i) {
        out_image.push_back(output_image[i]);
        if((i+1) % 3 == 0) {
            out_image.push_back(255);
        }
    }
    
    // Output the data
    error = lodepng::encode(output_file, out_image, width, height);

    //if there's an error, display it
    if(error) std::cout << "encoder error " << error << ": "<< lodepng_error_text(error) << std::endl;

    delete[] input_image;
    delete[] output_image;
    return 0;

}